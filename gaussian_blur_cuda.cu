
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <complex.h>
#include <math.h>
#include <sys/mman.h>

#define DIV_ROUND_UP(n, d)  (((n) + (d) - 1) / (d))

#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)
inline void _cuda_check(hipError_t ret, const char *file, int line)
{
	if (ret != hipSuccess) {
		fprintf(stderr, "CudaErr: %s (%s:%d)\n", hipGetErrorString(ret), file, line);
		exit(1);
	}
}

void read_image(char **Input, size_t *width, size_t *height, size_t *max, unsigned char **pixels)
{
    FILE *input_fd = fopen(*Input, "r+");
    if (!input_fd)
    {
        fprintf(stderr, "file does not exist");
        exit(1);
    }
    fseek(input_fd, 3, SEEK_CUR); //skip P5\n
    //printf("current line pos: %lu\n", ftell(input_fd));
    char c;
    if (fscanf(input_fd, " %lu %lu %lu%c", width, height, max, &c) != 4)
    {
        fprintf(stderr, "Info reading error\n");
        exit(1);
    }
    //printf("current line pos: %lu\n", ftell(input_fd));
    // fprintf(stdout, "width %lu, height %lu, max %lu, c%d\n", *width, *height, *max, c == '\n');

    //read the image into memory
    *pixels = (unsigned char*)calloc((*width) * (*height), sizeof(unsigned char));
    if (fread(*pixels, sizeof(unsigned char), (*width) * (*height), input_fd) != (*width) * (*height))
    {
        fprintf(stderr, "Image reading error\n");
        exit(1);
    }

    fclose(input_fd);
}

// Write pgm format
void write_image(unsigned char *g_map, size_t width, size_t height, size_t max, char *Output)
{
    char *info = (char*)calloc(100, sizeof(char));
    sprintf(info, "P5\n%lu %lu\n%lu\n", width, height, max);
    FILE *output_fd = fopen(Output, "w");
    fwrite(info, sizeof(char), strlen(info), output_fd);
    fwrite(g_map, sizeof(unsigned char), width * height, output_fd);
    fprintf(output_fd, "\n");
    fclose(output_fd);
}

float *gaussian_blur_matrix(size_t order, size_t sigma)
{

    float *matrix = (float*)calloc(order * order, sizeof(float));
    size_t x_0, y_0;
    x_0 = order / 2;
    y_0 = x_0;
    for (size_t y = 0; y < order; y++)
    {
        for (size_t x = 0; x < order; x++)
        {
            float x_dis = (float)x_0 - (float)x;
            float y_dis = (float)y_0 - (float)y;
            //printf("x_dis %f 7_dis %f\n", x_dis, y_dis);
            matrix[y * order + x] = expf((-1) * (x_dis * x_dis + y_dis * y_dis) / (2 * sigma * sigma));
        }
    }
    return matrix;
}

//__shared__ width, height, order

__device__ float blur_kernel_old(unsigned char* pixels, long width, long height, long x, long y, long order, float* mat){
    long start_x = x - order / 2;
    long start_y = y - order / 2;
    long curr_x, curr_y;
    float val = 0;
    for (long j = 0; j < order; j++)
    {
        curr_y = start_y + j;
        for (long i = 0; i < order; i++)
        {
            curr_x = start_x + i;

            if ((curr_x < 0 || curr_x >= width) && (curr_y < 0 || curr_y >= height))
            {
                if (curr_x < 0)
                    curr_x = 0;
                else if (curr_x >= width)
                    curr_x = width - 1;
                if (curr_y < 0)
                    curr_y = 0;
                else if (curr_y >= height)
                    curr_y = height - 1;
            }
            else if (curr_x < 0 || curr_x >= width)
            {
                if (curr_x < 0)
                    curr_x = 0;
                else if (curr_x >= width)
                    curr_x = width - 1;
            }
            else if (curr_y < 0 || curr_y >= height)
            {
                if (curr_y < 0)
                    curr_y = 0;
                else if (curr_y >= height)
                    curr_y = height - 1;
            }
            val += pixels[curr_y * width + curr_x] * mat[j * order + i];
        }
    }
    return val;
}

__device__ float blur_kernel(unsigned char* pixels, long width, long height, long x, long y, long order, float* mat, float k){
    long start_x = x - order / 2;
    long start_y = y - order / 2;
    long curr_x, curr_y;
    float val = 0;
    for (long j = 0; j < order; j++)
    {
        curr_y = start_y + j;
        if (curr_y < 0)
            curr_y = 0;

        else if (curr_y >= height)
            curr_y = height - 1;

        long pixel_row = curr_y * width;
        long mat_row = j * order;

        for (long i = 0; i < order; i++)
        {
            curr_x = start_x + i;

            if (curr_x < 0)
                curr_x = 0;
                
            else if (curr_x >= width)
                curr_x = width - 1;

            val += pixels[pixel_row + curr_x] * mat[mat_row + i];
        }
    }
    val /= k;
    return val;
}

__global__ void gaussian_blur_kernel_old(unsigned char *pixels, float *mat, unsigned char *output, long width, long height, float max, long order, float k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("row %d, col %d\n" row, col);
    // Discard out of bound coordinates
    if(row >= height || col >= width)
        return;

    //switched row and col
    float val = blur_kernel(pixels, width, height, col, row, order, mat, k);
    output[row * width + col] = (unsigned char)(val);
}

__global__ void gaussian_blur_kernel(unsigned char *pixels, float *mat, unsigned char *output, long width, long height, float max, long order, float k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Discard out of bound coordinates
    if(row >= height || col >= width)
        return;

    long start_y = row - order / 2;
    long start_x = col - order / 2;
    long curr_x, curr_y;
    float val = 0;
    for (long j = 0; j < order; j++)
    {
        curr_y = start_y + j;
        if (curr_y < 0)
            curr_y = 0;

        else if (curr_y >= height)
            curr_y = height - 1;

        long pixel_row = curr_y * width;
        long mat_row = j * order;

        for (long i = 0; i < order; i++)
        {
            curr_x = start_x + i;

            if (curr_x < 0)
                curr_x = 0;

            else if (curr_x >= width)
                curr_x = width - 1;

            val += pixels[pixel_row + curr_x] * mat[mat_row + i];
        }
    }
    val /= k;
    output[row * width + col] = (unsigned char)(val);
}



unsigned char *gaussian_blur_apply_cuda(unsigned char *pixels, long width, long height, float sigma, float max)
{
    // long order = (sigma * 6 % 2) == 0 ? sigma * 6 + 1 : sigma * 6;

    unsigned char *out_pixels_device, *out_pixels, *pixels_device;
    float *mat_device;

    //new
    long order = (long)(ceilf(sigma * 6));
    order = order % 2 == 0 ? order + 1 : order;
    // printf("order: %long\n", order);

    // Allocate needed matrices locally
    float *mat = gaussian_blur_matrix(order, sigma); //__shared__
    out_pixels = (unsigned char*)calloc(width * height, sizeof(unsigned char));

    //new
    float k = (2 * M_PI * sigma * sigma);
    
    int image_size = width * height * sizeof(unsigned char);
    
    //  allocate memory on device for needed matrices
    cuda_check(hipMalloc(&out_pixels_device, image_size));
    cuda_check(hipMalloc(&pixels_device, image_size));
    cuda_check(hipMalloc(&mat_device, order * order * sizeof(float)));

    // Copy data onto device
    cuda_check(hipMemcpy(pixels_device, pixels, image_size, hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(mat_device, mat, order * order * sizeof(float), hipMemcpyHostToDevice));

    //Invoke kernel function
    dim3 block_dim(32, 32);
    dim3 grid_dim(DIV_ROUND_UP(width, block_dim.x), DIV_ROUND_UP(height, block_dim.y));
    
    //  Catch errors
    gaussian_blur_kernel_old<<<grid_dim, block_dim>>>(pixels_device, mat_device, out_pixels_device, width, height, max, order, k);
    cuda_check(hipPeekAtLastError());      /* Catch configuration errors */
	cuda_check(hipDeviceSynchronize());    /* Catch execution errors */

    //  Copy output from device back to host
    cuda_check(hipMemcpy(out_pixels, out_pixels_device, image_size, hipMemcpyDeviceToHost));

    // Free memory on device
    cuda_check(hipFree(out_pixels_device));
	cuda_check(hipFree(mat_device));
	cuda_check(hipFree(pixels_device));

    // unsigned char *out_pixels = calloc(width * height, sizeof(unsigned char));

    free(mat);
    return out_pixels;
}


int main(int argc, char *argv[])
{
    char *Input = (char*)calloc(100, sizeof(char));
    unsigned char *pixels = NULL;
    unsigned char *out_pixels = NULL;
    char *Output = (char*)calloc(100, sizeof(char));
    float sigma;

    size_t width, height, max;
    if (argc != 4)
    {
        fprintf(stderr,
                "Usage: ./mandelbrot_serial order xcenter ycenter zoom cutoff\n");
        exit(1);
    }
    sscanf(argv[1], " %s", Input);
    sscanf(argv[2], " %s ", Output);
    sscanf(argv[3], " %f ", &sigma);
    //fprintf(stdout, "%s %s %lu\n", Input, Output, sigma);
    //read in original image
    read_image(&Input, &width, &height, &max, &pixels);

    //gaussian_blur the image
    out_pixels = gaussian_blur_apply_cuda(pixels, (long)width, (long)height, sigma, (float)max);
    //write out the final image
    write_image(out_pixels, width, height, max, Output); //writes the unprocessed image

    free(Input);
    free(Output);
    free(pixels);
    return 0;
}